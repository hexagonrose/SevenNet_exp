#include "hip/hip_runtime.h"
#include "sptp.hpp"
#include <cmath>

#define MAX_IR 11 // up to L_max =5
#define MAX_IN2 36 // up to L_max =5
#define WARPSIZE 32 
#define MAX_IN1_IR_CNT 32 // not just len(i_in1) but if u is larger than 32 need multiple IR_CNT 
#define MAX_NUM_PATH 512 // also need to account for u > 32
#define MAX_U_FIBER_CNT 5265 // up to L_max =5
#define MAX_U_CG_VAL_CNT 344 // up to L_max =5

// struct __align__(8) path_struct {
//     const u_short k_start;
//     const u_short k_end;  
//     const u_char k_val;
//     const u_char j_start;
//     const u_char j_val;
//     const u_char j_end;
// };

// struct __align__(4) fiber_struct {
//     const u_char i_idx;
//     const u_char j_idx;
//     const u_char k_idx;
//     const u_char cg_idx;
// };

// struct __align__(4) fiber_idx_struct {
//     const u_short start_idx;
//     const u_short end_idx;  
// };

__constant__ int in1_idxing[MAX_IN1_IR_CNT];
__constant__ int in1_ival[MAX_IN1_IR_CNT];
__constant__ int in1_related_path_idx[MAX_IN1_IR_CNT];

__constant__ ushort2 path_array1[MAX_NUM_PATH];
__constant__ uchar4 path_array2[MAX_NUM_PATH];
__constant__ ushort2 per_path_fiber_start[MAX_NUM_PATH];
__constant__ float path_weight[MAX_NUM_PATH];
__constant__ int per_path_weight_pos[MAX_NUM_PATH];

__constant__ uchar4 fiber_array[MAX_U_FIBER_CNT];
__constant__ float unique_cg_val[MAX_U_CG_VAL_CNT];

template <typename scalar_t>
__global__ void fwd_sptp_lienar_kernel_v2_shared(
    const float* __restrict__ in1,
    const float* __restrict__ in2,
    const float* __restrict__ weight,
    float* __restrict__ out,

    const size_t batch_size,
    const size_t out_size,
    const size_t weight_size,
    const size_t in1_size,
    const size_t in2_size,
    const size_t path_cnt,
    const size_t max_ir_dim
    )
    {
    extern __shared__ scalar_t shmem[];
    // Input dL_dO => batch, ir, mul order
    // 2D grid, 2D block
    // grid (path, batch), block (mul(same path), batch)
    // intra-warp (u parallel) x , inter-warp (batch) y 
    const int target_in1 = blockIdx.x;
    const int global_t_batch_idx = blockIdx.y * blockDim.y + threadIdx.y;
    const int shmem_warp_pos_start = threadIdx.y*blockDim.x;

    if(global_t_batch_idx >= batch_size) return;

    // const int t_g_edge_dest = ;
    // const int t_g_edge_src = ;

    // check given path (path per thread_block)

    // start_end of out for a block
    // divide by u 

    // load all in2 to shmem
    // load all nnz fiber to shmem
    // load cg value (to register?)
    // load all w 
    // sync
    
    // no init needed just copy

    scalar_t* my_batch_shmem_start = shmem + threadIdx.y * (blockDim.x * (max_ir_dim*3) );

    scalar_t* my_shmem_in1 = my_batch_shmem_start + threadIdx.x*max_ir_dim; 
    scalar_t* my_shmem_uvuv = my_batch_shmem_start +  blockDim.x*(max_ir_dim) + threadIdx.x*max_ir_dim;
    scalar_t* shmem_scratch = my_batch_shmem_start + blockDim.x*(max_ir_dim*2);
    
    scalar_t* shmem_in2 = shmem + blockDim.y * (blockDim.x * (max_ir_dim*3)) + threadIdx.y * in2_size;


    // dL_dO size : WARPSIZE * MAX_IR (all warps) * concurrent_batch (warp cnt)
    // dL_dO size : out_size 
    // (which i_in1 path, batch) (mul, batch)
    // need a lot of register.. (unless i make macro for all cases)

    // what defines the target_in1 ?? that is the question need z axis?

    // load part of in1 from main mem
    // in1 (z, mul, ir)
    const int in1_start = in1_idxing[target_in1];
    const int in1_end = in1_idxing[target_in1+1];
    const int i_val = in1_ival[target_in1];
    const int path_idx_start = in1_related_path_idx[target_in1];
    const int path_idx_end = in1_related_path_idx[target_in1+1];

    // using reg_dL_din1 for dummy => need to initialize ...
    for(int shmem_idx = threadIdx.x, in1_idx = global_t_batch_idx*in1_size + in1_start+threadIdx.x; in1_idx < global_t_batch_idx*in1_size + in1_end; shmem_idx+=WARPSIZE, in1_idx+=WARPSIZE) {
        shmem_scratch[shmem_idx] = in1[in1_idx];
    }
    __syncwarp();
    for(int i =0, shmem_idx = threadIdx.x*i_val; i<i_val; i++, shmem_idx++){
        my_shmem_in1[i] = shmem_scratch[shmem_idx];
    }

    // load all in2 from main mem
    for(int shmem_idx = threadIdx.x, in2_idx = global_t_batch_idx*in2_size + threadIdx.x; shmem_idx < in2_size; in2_idx+=WARPSIZE, shmem_idx+=WARPSIZE) {
        shmem_in2[shmem_idx] = in2[in2_idx];
    }
    __syncwarp();

    // for path_chunk
    // path idx == k idx
    // path index == 
    const int g_t_dldo_start = global_t_batch_idx*out_size;
    const int g_t_w_start = global_t_batch_idx*weight_size;

    for(int path_idx=path_idx_start; path_idx < path_idx_end; path_idx++){
        const ushort2 path_info1 = path_array1[path_idx]; // k_start, k_end
        const uchar4 path_info2 = path_array2[path_idx]; // k_val, j_start, j_val, j_end

        for(int i=0; i<max_ir_dim;i++){
            my_shmem_uvuv[i] = 0.0;
        }
        
        // stall due to global memory access (better if it is load to shared memory and accessed)
        // possible optimization point with gather scatter      
        // odd number of k_val (2n+1) no bank conflict
        
        // Loading Weight from global memory is a major memory bottleneck
        const int weight_pos = g_t_w_start + per_path_weight_pos[path_idx]+threadIdx.x;
        float reg_w_path_norm = weight[weight_pos] * path_weight[path_idx];
        
        const ushort2 fiber_idx_info = per_path_fiber_start[path_idx];
        // for nnz in the fiber
        // uchar4 fiber;
        for(ushort fiber_idx = fiber_idx_info.x; fiber_idx < fiber_idx_info.y; fiber_idx++){
            // mult k with all w => dL_duvuv
            uchar4 fiber = fiber_array[fiber_idx]; // i, j, k, cg idx
            my_shmem_uvuv[fiber.z] += my_shmem_in1[fiber.x] * shmem_in2[path_info2.y+fiber.y] * unique_cg_val[fiber.w];
        }
        // store out first in shared mem
        for(int i =0, shmem_idx = threadIdx.x*path_info2.x; i<path_info2.x; i++, shmem_idx++){
            shmem_scratch[shmem_idx] = my_shmem_uvuv[i] * reg_w_path_norm;
        }
        __syncwarp();
        // store out in main mem
        for(int shmem_idx = threadIdx.x, in1_idx = g_t_dldo_start+path_info1.x + threadIdx.x; in1_idx < g_t_dldo_start+ path_info1.y; in1_idx+=WARPSIZE, shmem_idx+=WARPSIZE) {
            out[in1_idx] = shmem_scratch[shmem_idx];
        }

        // for (int i =0, dldo_idx = g_t_dldo_start + path_info1.x + threadIdx.x*path_info2.x; i<path_info2.x; i++, dldo_idx++){
        //     out[dldo_idx] = my_shmem_uvuv[i] * reg_w_path_norm;
        // }
    }
}


void fwd_sptp_linear_cuda_v2_shared(
    torch::Tensor in1, 
    torch::Tensor in2,
    torch::Tensor weight,
    torch::Tensor out,

    torch::Tensor t_in1_idxing,
    torch::Tensor t_in1_ival,
    torch::Tensor t_in1_related_path_idx,

    torch::Tensor t_path_array1,
    torch::Tensor t_path_array2,
    torch::Tensor t_per_path_fiber_start,
    torch::Tensor t_path_weight,
    torch::Tensor t_per_path_weight_pos,

    torch::Tensor t_fiber_array,
    torch::Tensor t_unique_cg_val,

    size_t path_cnt,
    size_t per_block_batch,
    size_t max_ir_dim
    ){

    // TODO: not transposed (z, mul, ir)
    const auto batch_size = in1.size(0);
    const auto in1_size = in1.size(1);
    const auto in2_size = in2.size(1);
    const auto out_size = out.size(1);
    const auto weight_size = weight.size(1);

    dim3 grid(path_cnt, batch_size/per_block_batch);
    dim3 block(WARPSIZE, per_block_batch);

    // setup constant memory 
    hipMemcpyToSymbol(HIP_SYMBOL(in1_idxing), t_in1_idxing.data<int>(), at::numel(t_in1_idxing)*sizeof(int)); // int , MAX_IN1_IR_CNT
    hipMemcpyToSymbol(HIP_SYMBOL(in1_ival), t_in1_ival.data<int>(),  at::numel(t_in1_ival)*sizeof(int)); // int , MAX_IN1_IR_CNT
    hipMemcpyToSymbol(HIP_SYMBOL(in1_related_path_idx), t_in1_related_path_idx.data<int>(), at::numel(t_in1_related_path_idx)*sizeof(int)); // int  , MAX_IN1_IR_CNT
    
    hipMemcpyToSymbol(HIP_SYMBOL(path_array1), t_path_array1.data<u_short>(), at::numel(t_path_array1)*sizeof(u_short) ); // ushort2, MAX_NUM_PATH
    hipMemcpyToSymbol(HIP_SYMBOL(path_array2), t_path_array2.data<u_char>(), at::numel(t_path_array2)*sizeof(u_char)); // uchar4, MAX_NUM_PAT
    hipMemcpyToSymbol(HIP_SYMBOL(per_path_fiber_start), t_per_path_fiber_start.data<u_short>(), at::numel(t_per_path_fiber_start)*sizeof(u_short)); // ushort2, MAX_NUM_PATH
    hipMemcpyToSymbol(HIP_SYMBOL(path_weight), t_path_weight.data<float>(), at::numel(t_path_weight)*sizeof(float)); // float, MAX_NUM_PATH
    hipMemcpyToSymbol(HIP_SYMBOL(per_path_weight_pos), t_per_path_weight_pos.data<int>(), at::numel(t_per_path_weight_pos)*sizeof(int)); // int , MAX_NUM_PATH

    hipMemcpyToSymbol(HIP_SYMBOL(fiber_array), t_fiber_array.data<u_char>(), at::numel(t_fiber_array)*sizeof(u_char)); // u_char4), MAX_U_FIBER_CNT 
    hipMemcpyToSymbol(HIP_SYMBOL(unique_cg_val), t_unique_cg_val.data<float>(), at::numel(t_unique_cg_val)*sizeof(float) ); // float , MAX_U_CG_VAL_CNT

    const int shared_memory_bytes = sizeof(float) * per_block_batch * (WARPSIZE * (max_ir_dim*3) + in2_size);

    // int carveout = 100;
    // CHECK_CUDA_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
    //     sptp_all_forward_kernel_v1<float>),
    //     hipFuncAttributePreferredSharedMemoryCarveout, carveout));

    CHECK_CUDA_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
        fwd_sptp_lienar_kernel_v2_shared<float>),
        hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory_bytes));

    fwd_sptp_lienar_kernel_v2_shared<float><<<grid, block, shared_memory_bytes>>>(
        in1.data<float>(),
        in2.data<float>(),
        weight.data<float>(),
        out.data<float>(),
        
        batch_size,
        out_size,
        weight_size,
        in1_size,
        in2_size,
        path_cnt,
        max_ir_dim
        );
}
    